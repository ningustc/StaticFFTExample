
#include <hip/hip_runtime.h>
#ifdef CALLBACK
int callback_main(int);
#else
int no_callback_main(int);
#endif
int main(int argc, const char **argv)
{
    int device = argc > 1 ? atoi(argv[1]) : 0;
#ifdef CALLBACK
    callback_main(device);
#else
    no_callback_main(device);
#endif
    return 0;
}
